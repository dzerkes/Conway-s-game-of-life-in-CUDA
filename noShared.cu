#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ void t(int* world, int* state, int worldX, int worldY, int* a)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;


	if (row < worldY && col < worldX){
		int pos = row * worldX + col;
		int tmp = 0;

		for (int off_row = row - 5; off_row <= row + 5; off_row++){
			for (int off_col = col - 5; off_col <= col + 5; off_col++)
				if (!(off_row < 0 || off_row >= worldY || off_col < 0 || off_col >= worldX || (off_row == row && off_col == col))) //or substract itself
					tmp += world[off_row*worldX + off_col];//tmp = tmp;				
		}

		//cout << world[row * worldX + col] << " | ";

		if (tmp >= 34 && tmp <= 58){
			if (tmp <= 45 && state[pos] == 0){
				state[pos] = 1;
				atomicAdd(a, 1);
			}
		}
		else {
			if (state[pos] == 1){
				state[pos] = 0;
				atomicSub(a, 1);
			}
		}

		
	}
}

int main()
{
    
	int worldX;
	int worldY;
	int alive = 0;
	int* d_alive;

	printf("Please enter the width of the array : ");
	scanf("%d", &worldX);

	printf("Please enter the height of the array : ");
	scanf("%d", &worldY);

	printf("worldX : %d, worldY : %d\n", worldX, worldY);


    // Add vectors in parallel.
	//hipError_t cudaStatus;
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

	int population = worldX * worldY;
	int* world = (int*)malloc(sizeof(int*) * population);

	srand(time(NULL));
	for (int i = 0; i < worldX; i++)
	{
		for (int j = 0; j < worldY; j++)
		{
			if ((i + j) % 2 == 0) {
				world[i*worldX + j] = 1;
			}
			else {
				world[i*worldX + j] = 0;
			}
			//printf("%d ", c[i*arraySizex + j]);
			if (world[i*worldX + j] == 1)
			{
				alive++;
			}
		}
		//printf("\n");
	}

	printf("%d", alive);
	printf("\n");


	//for (int row = 0; row < worldY; row++){
	//	printf("| ");
	//	for (int col = 0; col < worldX; col++){
	//		printf("%d | ", world[row * worldX + col]);

	//	}
	//	printf("\n ");
	//}

	printf("\n");

	hipMalloc((void**)&d_alive, sizeof(int));
	hipMemcpy(d_alive, &alive, sizeof(int), hipMemcpyHostToDevice);

	int gridx = 16;
	int gridy = 16;
	dim3 grid((worldX / gridx) + 1, (worldY / gridy) + 1);
	dim3 blockSize(gridx, gridy);

	int* d_world;
	int* state;
	size_t size = sizeof(int*) * population;
	hipMalloc(&d_world, size);
	hipMemcpy(d_world, world, size, hipMemcpyHostToDevice);
	hipMalloc(&state, size);
	hipMemcpy(state, world, size, hipMemcpyHostToDevice);

	while (alive > 0)
	{
		t << < grid, blockSize >> >(d_world, state, worldX, worldY, d_alive);
		hipMemcpy(world, state, size, hipMemcpyDeviceToHost);
		hipMemcpy(d_world, state, size, hipMemcpyDeviceToDevice);
		hipMemcpy(&alive, d_alive, sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		//for (int row = 0; row < worldY; row++){
		//	printf("| ");
		//	for (int col = 0; col < worldX; col++){
		//		printf("%d | ", test[row * worldX + col]);

		//	}
		//	printf("\n ");
		//}

		printf("%d\n", alive);
	}
	
	hipFree(d_world);
	free(world);
	return 0;
}
