
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;



void find_alive(){

}


int main(){

	int worldX, worldY;

	printf("Please enter the width of the array : ");
	scanf("%d", &worldX);

	printf("Please enter the height of the array : ");
	scanf("%d", &worldY);

	int population = worldX * worldY;
	int* world = (int*)malloc(sizeof(int*) * population);
	int* count = (int*)malloc(sizeof(int*) * population);
	int* state = (int*)malloc(sizeof(int*) * population);
	int alive = 0;

	// Random initial polulation
	srand(time(NULL));
	for (int row = 0; row < worldY; row++)
		for (int col = 0; col < worldX; col++){
			int rand_val = rand() % 2;
			world[row*worldX + col] = rand_val;
			if (rand_val) alive += 1;
		}
			
	cout << alive;
	memcpy(state, world, sizeof(int*) * population);
	int lowest = INT_MAX;
	while (alive > 0){
		
		// Calculate alive neighbours and print polulation
		for (int row = 0; row < worldY; row++){
			//cout << "| ";
			int tmp = 0;
			for (int col = 0; col < worldX; col++){

				for (int off_row = row - 5; off_row <= row + 5; off_row++){
					for (int off_col = col - 5; off_col <= col + 5; off_col++)
						if (!(off_row < 0 || off_row >= worldY || off_col < 0 || off_col >= worldX || (off_row == row && off_col == col))) //or substract itself
							tmp += world[off_row*worldX + off_col];//tmp = tmp;				
				}

				//cout << world[row * worldX + col] << " | ";
				count[row * worldX + col] = tmp;
				if (tmp >= 34 && tmp <= 58){
					if (tmp <= 45 && state[row * worldX + col] == 0){
						state[row * worldX + col] = 1;
						alive += 1;
					}

				}
				else {
					if (state[row * worldX + col] == 1){
						state[row * worldX + col] = 0;
						alive -= 1;
					}
					
				}
				tmp = 0;
			}

			//cout << "\n";
		}

		//cout << "\n\n\n";

		//for (int row = 0; row < worldY; row++){
		//	cout << "| ";
		//	for (int col = 0; col < worldX; col++){
		//		cout << count[row * worldX + col] << " | ";
		//	}
		//	cout << "\n";
		//}


		//cout << "\n\n\n";

		memcpy(world, state, sizeof(int*) * population);
		//for (int row = 0; row < worldY; row++){
		//	cout << "| ";
		//	for (int col = 0; col < worldX; col++){
		//		cout << world[row * worldX + col] << " | ";

		//	}
		//	cout << "\n";
		//}
		if (alive < lowest){
			cout << '\n' << alive;
			lowest = alive;
		}
	}

	
	
	






	return 0;
}
