#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
hipError_t addWithCuda(int *c, int width, int height, int size, int alive);

__global__ void addKernel(int* c, int* n, int width, int height, int* alive)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	if (row < width && col < width)
	{
		//Shared
		const int block = 42;
		__shared__ int temp[block][block];

		int x = threadIdx.y;
		int y = threadIdx.x;

		if (x < 32 && y < 32) {
			temp[x + 5][y + 5] = c[row*width + col];
			//gemisma shared
			if (x < 5)
			{
				if (row >= 5)
				{
					temp[x][y + 5] = c[(row - 5) * width + col];
				}
				else 
				{
					temp[x][y + 5] = 0;
				}
				if (y < 5  )
				{
					if ((row >= 5 && col >= 5))
					{
						temp[x][y] = c[(row - 5) * width + (col - 5)];
					}
					else 
					{
						temp[x][y] =0;
					}

				}
				else if (y >= 27)
				{
					if (row >= 5 && (col + 5) < width)
					{
						temp[x][y + 10] = c[(row - 5)* width + (col + 5)];
					}
					else {
						temp[x][y + 10] = 0;
					}

				}

			}
			else if (x >= 27 )
			{
				if ((row + 5) < width)
				{
					temp[x + 10][y + 5] = c[(row + 5) * width + col];
				}
				else {
					temp[x + 10][y + 5] = 0;
				}
				if (y < 5)
				{
					if (col >= 5 && row + 5 < width)
					{
						temp[x + 10][y] = c[(row + 5) * width + (col - 5)];
					}
					else {
						temp[x + 10][y] = 0;
					}

				}
				else if (y >= 27)
				{
					if (row + 5 < width && (col + 5) < width)
					{
						temp[x + 10][y + 10] = c[(row + 5) * width + (col + 5)];
					}
					else {
						temp[x + 10][y + 10] = 0;
					}

				}


			}

			if (y < 5 )
			{
				if (col >= 5) {
					temp[x + 5][y] = c[row*width + (col - 5)];
				}
				else {
					temp[x + 5][y] = 0;
				}
			}
			else if (y >= 27 )
			{
				if (col + 5 < width)
				{
					temp[x + 5][y + 10] = c[row*width + (col + 5)];
				}
				else {
					temp[x + 5][y + 10] = 0;
				}
			}


			//telos gemismatos
			__syncthreads();
			int alive_n = 0;
			int posx = x + 5;
			int posy = y + 5;
			for (int i = posx - 5; i <= posx + 5; i++)
			{
				for (int j = posy - 5; j <= posy + 5; j++)
				{
					if (i >= 0 && i < block && j >= 0 && j < block && !(i == posx && j == posy))
					{
						alive_n += temp[i][j];
					}
				}
			}
			if (alive_n >= 34 && alive_n <= 58)
			{
				if (alive_n <= 45 && temp[posx][posy] == 0)
				{
					n[row*width + col] = 1;
					atomicAdd(alive, 1);
				}
			}
			else
			{
				if (temp[posx][posy] == 1)
				{
					n[row*width + col] = 0;
					atomicSub(alive, 1);
				}
			}
		}
	}
}
int main()
{

	int arraySizex = 1024;
	int arraySizey = 1024;
	int alive = 0;
	int size = arraySizex * arraySizey;
	int* c = (int*)malloc(sizeof(int*) * size);

	for (int i = 0; i < arraySizex; i++)
	{
		for (int j = 0; j < arraySizey; j++)
		{
			if ((i + j) % 2 == 0) {
				c[i*arraySizex + j] = 1;
			}
			else {
				c[i*arraySizex + j] = 0;
			}
			//printf("%d ", c[i*arraySizex + j]);
			if (c[i*arraySizex + j] == 1)
			{
				alive++;
			}
		}
		//printf("\n");
	}
	printf("1 generation alive :%d and dead :%d \n", alive, size - alive);

	hipError_t cudaStatus = addWithCuda(c, arraySizex, arraySizey, size, alive);
	cudaStatus = hipDeviceReset();
	printf("Press Any Key to Continue\n");
	getchar();
	free(c);
	return 0;
}


hipError_t addWithCuda(int *c, int width, int height, int size, int alive)
{
	clock_t tic = clock();
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	hipError_t cudaStatus;

	int* dev_c;
	int* dev_n;
	int* dev_alive;

	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_alive, sizeof(int));
	cudaStatus = hipMemcpy(dev_alive, &alive, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMalloc(&dev_c, size * sizeof(int));
	cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMalloc(&dev_n, size * sizeof(int));
	cudaStatus = hipMemcpy(dev_n, c, size * sizeof(int), hipMemcpyHostToDevice);

	
	
	int generation = 2;
	int next_alive = 0;
	while (alive > 0 && (next_alive != alive))
	{
		next_alive = alive;
		addKernel << < numBlocks, threadsPerBlock >> > (dev_c, dev_n, width, height, dev_alive);
		cudaStatus = hipMemcpy(&alive, dev_alive, sizeof(int), hipMemcpyDeviceToHost);
		cudaStatus = hipMemcpy(c, dev_n, size * sizeof(int), hipMemcpyDeviceToHost);
		cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipDeviceSynchronize();


		//printf("%d generation : \n",generation);
		//for (int row = 0; row < width; row++){
		//for (int col = 0; col < height; col++){
		//printf("%d ", c[row * width + col]);

		//}
		//printf("\n");
		//}

		printf("%d generation alive :%d and dead:%d \n", generation, alive, size - alive);
		generation++;

	}
	clock_t toc = clock();

	printf("%d generations time elapsed: %f seconds\n",generation-1, (double)(toc - tic) / CLOCKS_PER_SEC);
	hipFree(dev_c);
	hipFree(dev_n);
	hipFree(dev_alive);
	return cudaStatus;
}
